#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <chrono>

#define RGB_COMPONENT_COLOR 255

#define BLUR_SIZE 5

using namespace std;

typedef struct {
	unsigned char red, green, blue;
} PPMPixel;

typedef struct {
	int x, y;
	PPMPixel *data;
} PPMImage;

unsigned char *readPPM(const char *filename, int &x, int &y)
{
	char buff[16];
	unsigned char* imgchar;
	FILE *fp;
	int c, rgb_comp_color;
	//open PPM file for reading
	fp = fopen(filename, "rb");
	if (!fp) {
		fprintf(stderr, "Unable to open file '%s'\n", filename);
		exit(1);
	}

	//read image format
	if (!fgets(buff, sizeof(buff), fp)) {
		perror(filename);
		exit(1);
	}

	//check the image format
	if (buff[0] != 'P' || buff[1] != '6') {
		fprintf(stderr, "Invalid image format (must be 'P6')\n");
		exit(1);
	}

	//read image size information
	if (fscanf(fp, "%d %d", &x, &y) != 2) {
		fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
		exit(1);
	}

	//read rgb component
	if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
		fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
		exit(1);
	}

	//check rgb component depth
	if (rgb_comp_color != RGB_COMPONENT_COLOR) {
		fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
		exit(1);
	}

	while (fgetc(fp) != '\n');
	//memory allocation for pixel data
	imgchar = (unsigned char*)malloc(3 * x*y * sizeof(char));

	//read pixel data from file
	fread(imgchar, 3 * x, y, fp);


	fclose(fp);
	return imgchar;
}

void writePPM(unsigned char * img, int x, int y)
{
	FILE *fp;
	//open file for output
	fp = fopen("C:\\UCSP\\2019-I\\AP\\Practice\\blur\\blur\\img_blur.ppm", "wb");
	if (!fp) {
		fprintf(stderr, "Unable to open file '%s'\n", "out");
		exit(1);
	}

	//write the header file
	//image format
	fprintf(fp, "P6\n");

	//image size
	fprintf(fp, "%d %d\n", x, y);

	// rgb component depth
	fprintf(fp, "%d\n", RGB_COMPONENT_COLOR);

	// pixel data
	fwrite(img, 3 * x, y, fp);
	fclose(fp);
}

unsigned char* readBMP(char* file_name, int &width, int &height) {
	FILE* img = fopen(file_name, "rb");
	unsigned char header[54];
	fread(header, sizeof(unsigned char), 54, img);
	width = *(int*)&header[18];
	height = *(int*)&header[22];
	int size = width * height * 3;
	unsigned char* r_img = (unsigned char*)malloc(size * sizeof(unsigned char));
	fread(r_img, sizeof(unsigned char), size, img);
	fclose(img);
	return r_img;
}

void writeBMP(unsigned char* img, int width, int height) {
	FILE* f_img;
	int f_size = 54 + 3 * width* height;
	unsigned char file_header[14] = { 'B','M', 0,0,0,0, 0,0, 0,0, 54,0,0,0 };
	unsigned char info_header[40] = { 40,0,0,0, 0,0,0,0, 0,0,0,0, 1,0, 24,0 };
	unsigned char pad[3] = { 0,0,0 };
	file_header[2] = (unsigned char)(f_size);
	file_header[3] = (unsigned char)(f_size >> 8);
	file_header[4] = (unsigned char)(f_size >> 16);
	file_header[5] = (unsigned char)(f_size >> 24);
	info_header[4] = (unsigned char)(width);
	info_header[5] = (unsigned char)(width >> 8);
	info_header[6] = (unsigned char)(width >> 16);
	info_header[7] = (unsigned char)(width >> 24);
	info_header[8] = (unsigned char)(height);
	info_header[9] = (unsigned char)(height >> 8);
	info_header[10] = (unsigned char)(height >> 16);
	info_header[11] = (unsigned char)(height >> 24);
	f_img = fopen("C:\\UCSP\\2019-I\\AP\\Practice\\blur\\blur\\img_blur.bmp", "wb");
	fwrite(file_header, 1, 14, f_img);
	fwrite(info_header, 1, 40, f_img);
	for (int i = height - 1; i >= 0; i--) {
		fwrite(img + (width * (height - i - 1) * 3), 3, width, f_img);
		fwrite(pad, 1, (4 - (width * 3) % 4) % 4, f_img);
	}
	free(img);
	fclose(f_img);
}

// Blur GPU
__global__
void blurKernel(unsigned char* out, unsigned char* in, int w, int h) {
	int Col = threadIdx.x + blockIdx.x * blockDim.x;
	int Row = threadIdx.y + blockIdx.y * blockDim.y;
	int Offset = Row * w + Col;
	if ((Col < w) && (Row < h)) {
		int pixValR = 0;
		int pixValG = 0;
		int pixValB = 0;
		int pixels = 0;
		// Get the average of the surrounding BLUR_SIZE x BLUR_SIZE box
		for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE; blurRow++) {
			for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE; blurCol++) {
				int curRow = Row + blurRow;
				int curCol = Col + blurCol;
				// Verify we have a valid image pixel
				if ((curRow > -1) && (curRow < h) && (curCol > -1) && (curCol < w)) {
					int curOffset = curRow * w + curCol;
					pixValR += in[curOffset * 3];
					pixValG += in[curOffset * 3 + 1];
					pixValB += in[curOffset * 3 + 2];
					pixels++; // Keep track of number of pixels in the avg
				}
			}
		}
		// Write our new pixel value out
		out[Offset * 3] = (unsigned char)(pixValR / pixels);
		out[Offset * 3 + 1] = (unsigned char)(pixValG / pixels);
		out[Offset * 3 + 2] = (unsigned char)(pixValB / pixels);
	}
}

int main() {
	unsigned char* h_img_in;
	unsigned char* h_img_out;
	unsigned char* d_img_in;
	unsigned char* d_img_out;

	int width = 0;
	int height = 0;

	char* img_name = "D:\\Documentos\\Semestre 2020-2\\Computación Paralela y Distribuida\\Tareas\\CUDA\\AP-master\\CUDA\\Blur";
	//char* img_name = "C:\\UCSP\\2019-I\\AP\\Practice\\blur\\blur\\lenna.ppm";

	h_img_in = readBMP(img_name, width, height);
	//h_img_in = readPPM(img_name, width, height);
	cout << "Ready img_in" << endl;
	int size_grey = (width * height * sizeof(unsigned char)) * 3;
	int size_rgb = (width * height * sizeof(unsigned char)) * 3;
	h_img_out = (unsigned char*)malloc(size_grey * sizeof(unsigned char));
	cout << "Ready img_out" << endl;

	hipMalloc(&d_img_in, size_rgb);
	hipMemcpy(d_img_in, h_img_in, size_rgb, hipMemcpyHostToDevice);
	hipMalloc(&d_img_out, size_grey);
	hipMemcpy(d_img_out, h_img_out, size_grey, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(width / 32.0), ceil(height / 32.0), 1);
	dim3 dimBlock(32, 32, 1);

	chrono::time_point<chrono::system_clock> GPU_Start, GPU_End;

	GPU_Start = chrono::system_clock::now();
	blurKernel <<< dimGrid, dimBlock >>> (d_img_out, d_img_in, width, height);
	GPU_End = chrono::system_clock::now();

	cout << "GPU: " << chrono::duration_cast<chrono::nanoseconds>(GPU_End - GPU_Start).count() << "ns." << endl;

	hipMemcpy(h_img_out, d_img_out, size_grey, hipMemcpyDeviceToHost);

	writeBMP(h_img_out, width, height);
	//writePPM(h_img_out, width, height);

	hipFree(d_img_in);
	hipFree(d_img_out);

	free(h_img_in);
	free(h_img_out);

	return 0;
}
