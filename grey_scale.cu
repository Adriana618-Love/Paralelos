#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <chrono>

#define RGB_COMPONENT_COLOR 255

#define CHANNELS 3

using namespace std;

typedef struct {
	unsigned char red, green, blue;
} PPMPixel;

typedef struct {
	int x, y;
	PPMPixel *data;
} PPMImage;

unsigned char *readPPM(const char *filename, int &x, int &y)
{
	char buff[16];
	unsigned char* imgchar;
	FILE *fp;
	int c, rgb_comp_color;
	//open PPM file for reading
	fp = fopen(filename, "rb");
	if (!fp) {
		fprintf(stderr, "Unable to open file '%s'\n", filename);
		exit(1);
	}

	//read image format
	if (!fgets(buff, sizeof(buff), fp)) {
		perror(filename);
		exit(1);
	}

	//check the image format
	if (buff[0] != 'P' || buff[1] != '6') {
		fprintf(stderr, "Invalid image format (must be 'P6')\n");
		exit(1);
	}

	//read image size information
	if (fscanf(fp, "%d %d", &x, &y) != 2) {
		fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
		exit(1);
	}

	//read rgb component
	if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
		fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
		exit(1);
	}

	//check rgb component depth
	if (rgb_comp_color != RGB_COMPONENT_COLOR) {
		fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
		exit(1);
	}

	while (fgetc(fp) != '\n');
	//memory allocation for pixel data
	imgchar = (unsigned char*)malloc(3 * x*y * sizeof(char));

	//read pixel data from file
	fread(imgchar, 3 * x, y, fp);


	fclose(fp);
	return imgchar;
}

void writePPM(unsigned char * img, int x, int y)
{
	FILE *fp;
	//open file for output
	fp = fopen("C:\\UCSP\\2019-I\\AP\\Practice\\grey_scale\\grey_scale\\img_grey.ppm", "wb");
	if (!fp) {
		fprintf(stderr, "Unable to open file '%s'\n", "out");
		exit(1);
	}

	//write the header file
	//image format
	fprintf(fp, "P6\n");

	//image size
	fprintf(fp, "%d %d\n", x, y);

	// rgb component depth
	fprintf(fp, "%d\n", RGB_COMPONENT_COLOR);

	// pixel data
	fwrite(img, 3 * x, y, fp);
	fclose(fp);
}

unsigned char* readBMP(char* file_name, int &width, int &height) {
	FILE* img = fopen(file_name, "rb");
	unsigned char header[54];
	fread(header, sizeof(unsigned char), 54, img);
	width = *(int*)&header[18];
	height = *(int*)&header[22];
	int size = width * height * 3;
	unsigned char* r_img = (unsigned char*)malloc(size * sizeof(unsigned char));
	fread(r_img, sizeof(unsigned char), size, img);
	fclose(img);
	return r_img;
}

void writeBMP(unsigned char* img, int width, int height) {
	FILE* f_img;
	int f_size = 54 + 3 * width* height;
	unsigned char file_header[14] = { 'B','M', 0,0,0,0, 0,0, 0,0, 54,0,0,0 };
	unsigned char info_header[40] = { 40,0,0,0, 0,0,0,0, 0,0,0,0, 1,0, 24,0 };
	unsigned char pad[3] = { 0,0,0 };
	file_header[2] = (unsigned char)(f_size);
	file_header[3] = (unsigned char)(f_size >> 8);
	file_header[4] = (unsigned char)(f_size >> 16);
	file_header[5] = (unsigned char)(f_size >> 24);
	info_header[4] = (unsigned char)(width);
	info_header[5] = (unsigned char)(width >> 8);
	info_header[6] = (unsigned char)(width >> 16);
	info_header[7] = (unsigned char)(width >> 24);
	info_header[8] = (unsigned char)(height);
	info_header[9] = (unsigned char)(height >> 8);
	info_header[10] = (unsigned char)(height >> 16);
	info_header[11] = (unsigned char)(height >> 24);
	f_img = fopen("C:\\UCSP\\2019-I\\AP\\Practice\\grey_scale\\grey_scale\\img_grey.bmp", "wb");
	fwrite(file_header, 1, 14, f_img);
	fwrite(info_header, 1, 40, f_img);
	for (int i = height - 1; i >= 0; i--) {
		fwrite(img + (width * (height - i - 1) * 3), 3, width, f_img);
		fwrite(pad, 1, (4 - (width * 3) % 4) % 4, f_img);
	}
	free(img);
	fclose(f_img);
}

// we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__
void colorToGreyscaleConversion(unsigned char* Pout, unsigned char* Pin, int width, int height) {
	int Col = threadIdx.x + blockIdx.x * blockDim.x;
	int Row = threadIdx.y + blockIdx.y * blockDim.y;
	if ((Col < width) && (Row < height)) {
		// get 1D coordinate for the grayscale image
		int greyOffset = Row*width + Col;
		// one can think of the RGB image having
		// CHANNEL times columns than the grayscale image
		int rgbOffset = greyOffset * CHANNELS;
		unsigned char r = Pin[rgbOffset];		// red value for pixel
		unsigned char g = Pin[rgbOffset + 1];	// green value for pixel
		unsigned char b = Pin[rgbOffset + 2];	// blue value for pixel
		// perform the rescaling and store it
		// We multiply by floating point constants
		unsigned char gs = 0.21f*r + 0.71f*g + 0.07f*b;
		Pout[rgbOffset] = gs;
		Pout[rgbOffset + 1] = gs;
		Pout[rgbOffset + 2] = gs;
	}
}

int main() {
	unsigned char* h_img_in;
	unsigned char* h_img_out;
	unsigned char* d_img_in;
	unsigned char* d_img_out;

	int width = 0;
	int height = 0;

	//char* img_name = "C:\\UCSP\\2019-I\\AP\\Practice\\grey_scale\\grey_scale\\lenna.bmp";
	char* img_name = "C:\\UCSP\\2019-I\\AP\\Practice\\grey_scale\\grey_scale\\lenna.ppm";
	
	//h_img_in = readBMP(img_name, width, height);
	h_img_in = readPPM(img_name, width, height);
	cout << "Ready img_in" << endl;
	int size_grey = (width * height * sizeof(unsigned char)) * 3;
	int size_rgb = (width * height * sizeof(unsigned char)) * 3;
	h_img_out = (unsigned char*)malloc(size_grey * sizeof(unsigned char));
	cout << "Ready img_out" << endl;

	hipMalloc(&d_img_in, size_rgb);
	hipMemcpy(d_img_in, h_img_in, size_rgb, hipMemcpyHostToDevice);
	hipMalloc(&d_img_out, size_grey);
	hipMemcpy(d_img_out, h_img_out, size_grey, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(width/32.0), ceil(height/32.0), 1);
	dim3 dimBlock(32, 32, 1);

	chrono::time_point<chrono::system_clock> GPU_Start, GPU_End;

	GPU_Start = chrono::system_clock::now();
	colorToGreyscaleConversion <<< dimGrid, dimBlock >>> (d_img_out, d_img_in, width, height);
	GPU_End = chrono::system_clock::now();

	cout << "GPU: " << chrono::duration_cast<chrono::nanoseconds>(GPU_End - GPU_Start).count() << "ns." << endl;

	hipMemcpy(h_img_out, d_img_out, size_grey, hipMemcpyDeviceToHost);

	//writeBMP(h_img_out, width, height);
	writePPM(h_img_out, width, height);

	hipFree(d_img_in);
	hipFree(d_img_out);
	
	free(h_img_in);
	free(h_img_out);
	
	return 0;
}
